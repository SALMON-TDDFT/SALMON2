
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <complex>
#include <math.h>
#include <hip/hip_complex.h>

extern "C" {
void update_kvector_nonlocalpt_kernel(
		// Output
		hipDoubleComplex* ppg_zekr_uV,
		// Input (size)
		const int ik_s,
		const int ik_e,
		const int natom,
		const int ppg_nlma,
		const int ppg_nps,
		const int ia_tbl_size,
		// Input (ptr)
		const int* const ppg_ia_tbl,
		const int* const ppg_mps,
		const double* const ppg_rxyz,
		const double* ppg_uv,
		const double* kAc
		) {
	for (int ik = ik_s - 1; ik < ik_e; ik++) {
		for (int ilma = 0; ilma < ppg_nlma; ilma++) {
			const int iatom = ppg_ia_tbl[ilma] - 1;
			for (int j = 0; j < ppg_mps[iatom]; j++) {
				const double x = ppg_rxyz[0 + j * (3) + iatom * (3 * ppg_nps)];
				const double y = ppg_rxyz[1 + j * (3) + iatom * (3 * ppg_nps)];
				const double z = ppg_rxyz[2 + j * (3) + iatom * (3 * ppg_nps)];

				const double theta = kAc[0 + 3 * (ik - (ik_s - 1))] * x +
					kAc[1 + 3 * (ik - (ik_s - 1))] * y +
					kAc[2 + 3 * (ik - (ik_s - 1))] * z;
				const hipDoubleComplex ekr_conj = {cos(theta), -sin(theta)};
				const double tmp = ppg_uv[j + ilma * (ppg_nps)];
				const hipDoubleComplex res = {ekr_conj.x * tmp, ekr_conj.y * tmp};
				ppg_zekr_uV[j + ilma * (ppg_nps) + ik * (ppg_nps * ppg_nlma)] = res;
			}
		}
	}
}
}
