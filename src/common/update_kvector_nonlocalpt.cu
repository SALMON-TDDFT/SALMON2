
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <complex>
#include <math.h>
#include <hip/hip_complex.h>

extern "C" {
static const unsigned block_size = 256;
__global__ void update_kvector_nonlocalpt_kernel(
		// Output
		hipDoubleComplex* ppg_zekr_uV,
		// Input (size)
		const int ik_s,
		const int ik_e,
		const int natom,
		const int ppg_nlma,
		const int ppg_nps,
		const int ia_tbl_size,
		// Input (ptr)
		const int* const ppg_ia_tbl,
		const int* const ppg_mps,
		const double* const ppg_rxyz,
		const double* ppg_uv,
		const double* kAc
		) {
	const unsigned ik = (blockIdx.x / ppg_nlma) + ik_s - 1;
	if (ik >= ik_e) return;
	const unsigned ilma = blockIdx.x % ppg_nlma;

	const int iatom = ppg_ia_tbl[ilma] - 1;
	const int j_end = ppg_mps[iatom];

	const double kAc_x = kAc[0 + 3 * (ik - (ik_s - 1))];
	const double kAc_y = kAc[1 + 3 * (ik - (ik_s - 1))];
	const double kAc_z = kAc[2 + 3 * (ik - (ik_s - 1))];

	for (int j_offset = 0; j_offset < j_end; j_offset += block_size) {
		const int j = j_offset + threadIdx.x;
		if (j >= j_end) return;
		const double x = ppg_rxyz[0 + j * (3) + iatom * (3 * ppg_nps)];
		const double y = ppg_rxyz[1 + j * (3) + iatom * (3 * ppg_nps)];
		const double z = ppg_rxyz[2 + j * (3) + iatom * (3 * ppg_nps)];

		const double theta = kAc_x * x +kAc_y * y + kAc_z * z;
		const hipDoubleComplex ekr_conj = {cos(theta), -sin(theta)};
		const double tmp = ppg_uv[j + ilma * (ppg_nps)];
		const hipDoubleComplex res = {ekr_conj.x * tmp, ekr_conj.y * tmp};
		ppg_zekr_uV[j + ilma * (ppg_nps) + ik * (ppg_nps * ppg_nlma)] = res;
	}
}
void update_kvector_nonlocalpt_core(
		// Output
		hipDoubleComplex* ppg_zekr_uV,
		// Input (size)
		const int ik_s,
		const int ik_e,
		const int natom,
		const int ppg_nlma,
		const int ppg_nps,
		const int ia_tbl_size,
		// Input (ptr)
		const int* const ppg_ia_tbl,
		const int* const ppg_mps,
		const double* const ppg_rxyz,
		const double* ppg_uv,
		const double* kAc
		) {
	dim3 grid_dim((ik_e - (ik_s - 1) + 1) * ppg_nlma);
	dim3 block_dim(block_size);
	update_kvector_nonlocalpt_kernel<<<grid_dim, block_dim>>>(
			ppg_zekr_uV,
			ik_s, ik_e,
			natom,
			ppg_nlma, ppg_nps,
			ia_tbl_size,
			ppg_ia_tbl, ppg_mps,
			ppg_rxyz, ppg_uv,
			kAc
			);
	hipDeviceSynchronize();
}
}
