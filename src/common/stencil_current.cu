#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <complex>
#include <hip/hip_complex.h>
#include <hipcub/hipcub.hpp>

extern "C" {
    static constexpr unsigned block_size = 128;
    static constexpr unsigned Nd = 4;
    static_assert(block_size != 0 && (block_size & (block_size-1)) == 0, "block_size must be 2^n");

    void stencil_current_core_cpu(const int ik_s, const int ik_e, const int io_s, const int io_e, const double* const vec_k, const double* const vec_Ac,
                                  const int* const is_array, const int* const ie_array, const int* const is, const int* const ie, const int* const idx, const int* const idy,
                                  const int* const idz, const double* const nabt, const int ispin, const int im, const int spin_len, const std::complex<double>* const psi, const double* const BT,
                                  const double* const rocc, const double* const wtk, double* const jx, double* const jy, double* const jz) {
        const int xlen = ie_array[0] - is_array[0] + 1;
        const int ylen = ie_array[1] - is_array[1] + 1;
        const int zlen = ie_array[2] - is_array[2] + 1;
        int psi_index = xlen*ylen*zlen*(ispin - 1) + xlen*ylen*zlen*spin_len*(ik_e - ik_s + 1)*(io_e - io_s + 1)*(im - 1);
        for(int ik = ik_s-1; ik < ik_e; ik++) {
            double kAc[3];
            for(int i = 0; i < 3; i++) {
                kAc[i] = vec_k[i+3*ik] + vec_Ac[i];
            }
            for(int io = io_s-1; io < io_e; io++) {
                double rtmp = 0.0;
                std::complex<double> xtmp(0.0, 0.0);
                std::complex<double> ytmp(0.0, 0.0);
                std::complex<double> ztmp(0.0, 0.0);
                std::complex<double> cpsi;
                for(int iz = is[2]-1; iz < ie[2]; iz++) {
                    for(int iy = is[1]-1; iy < ie[1]; iy++) {
                        for(int ix = is[0]-1; ix < ie[0]; ix++) {
                            rtmp = rtmp + std::abs(psi[psi_index + ix + iy*xlen + iz*xlen*ylen])*std::abs(psi[psi_index + ix + iy*xlen + iz*xlen*ylen]);
                            cpsi = std::conj(psi[psi_index + ix + iy*xlen + iz*xlen*ylen]);
                            xtmp = xtmp + nabt[0] * cpsi * (psi[psi_index + (idx[ix + 1] - 1) + iy*xlen + iz*xlen*ylen])
                                        + nabt[1] * cpsi * (psi[psi_index + (idx[ix + 2] - 1) + iy*xlen + iz*xlen*ylen])
                                        + nabt[2] * cpsi * (psi[psi_index + (idx[ix + 3] - 1) + iy*xlen + iz*xlen*ylen])
                                        + nabt[3] * cpsi * (psi[psi_index + (idx[ix + 4] - 1) + iy*xlen + iz*xlen*ylen]);

                            ytmp = ytmp + nabt[4] * cpsi * (psi[psi_index + ix + (idy[iy + 1] - 1)*xlen + iz*xlen*ylen])
                                        + nabt[5] * cpsi * (psi[psi_index + ix + (idy[iy + 2] - 1)*xlen + iz*xlen*ylen])
                                        + nabt[6] * cpsi * (psi[psi_index + ix + (idy[iy + 3] - 1)*xlen + iz*xlen*ylen])
                                        + nabt[7] * cpsi * (psi[psi_index + ix + (idy[iy + 4] - 1)*xlen + iz*xlen*ylen]);

                            ztmp = ztmp + nabt[8]  * cpsi * (psi[psi_index + ix + iy*xlen + (idz[iz + 1] - 1)*xlen*ylen])
                                        + nabt[9]  * cpsi * (psi[psi_index + ix + iy*xlen + (idz[iz + 2] - 1)*xlen*ylen])
                                        + nabt[10] * cpsi * (psi[psi_index + ix + iy*xlen + (idz[iz + 3] - 1)*xlen*ylen])
                                        + nabt[11] * cpsi * (psi[psi_index + ix + iy*xlen + (idz[iz + 4] - 1)*xlen*ylen]);
                        }
                    }
                }
                psi_index += xlen*ylen*zlen*spin_len;
                double wrk1[3], wrk2[3], wrk3[3], wrk4[3];
                for(int i = 0; i < 3; i++) {
                    wrk1[i] = kAc[i]*rtmp;
                }
                wrk2[0] = xtmp.imag()*static_cast<double>(2);
                wrk2[1] = ytmp.imag()*static_cast<double>(2);
                wrk2[2] = ztmp.imag()*static_cast<double>(2);

                for(int i = 0; i < 3; i++) {
                    wrk3[i] = BT[i]*wrk2[0]+BT[i + 3]*wrk2[1]+BT[i + 6]*wrk2[2];
                }
                for(int i = 0; i < 3; i++) {
                    wrk4[i] = (wrk1[i] + wrk3[i])*rocc[io + ik*(io_e - io_s + 1)]*wtk[ik];
                }
                *jx += wrk4[0];
                *jy += wrk4[1];
                *jz += wrk4[2];
            }
        }
    }

    __global__ void stencil_current_kernel(double* const d_res, const hipDoubleComplex* const psi_data, const int xlen,
                                           const int ylen, const int xsize, const int ysize, const int zsize, const int maxlen, const double* const nabt, const int* const idx, const int* const idy,
                                           const int* const idz, const double nabt0, const double nabt1, const double nabt2, const double nabt3, const double nabt4, const double nabt5, const double nabt6,
                                           const double nabt7, const double nabt8, const double nabt9, const double nabt10, const double nabt11) {
        const int tid = blockIdx.x * blockDim.x + threadIdx.x;
        const int threadId = threadIdx.x;
        if (tid >= maxlen) {
            return;
        }
        typedef hipcub::BlockReduce<double, block_size> BlockReduce;
        __shared__ typename BlockReduce::TempStorage temp_storage;
        const int ix = tid % xsize;
        const int iy = (tid/xsize) % ysize;
        const int iz = (tid/(xsize*ysize)) % zsize;
        hipDoubleComplex tmp;
        double val, block_sum;
        const hipDoubleComplex cpsi = hipConj(psi_data[ix + iy*xlen + iz*xlen*ylen]);
        tmp = psi_data[ix + iy*xlen + iz*xlen*ylen]; // psi_data[ix][iy][iz]
        const double psi_abs = hipCabs(tmp);
        val = psi_abs*psi_abs;
        __syncthreads();
        block_sum = BlockReduce(temp_storage).Sum(val);
        if(threadId == 0) {
             atomicAdd(&d_res[0], block_sum);
        }

        tmp = psi_data[(idx[ix + 1] - 1) + iy*xlen + iz*xlen*ylen];
        val = nabt0*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        tmp = psi_data[(idx[ix + 2] - 1) + iy*xlen + iz*xlen*ylen];
        val += nabt1*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        tmp = psi_data[(idx[ix + 3] - 1) + iy*xlen + iz*xlen*ylen];
        val += nabt2*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        tmp = psi_data[(idx[ix + 4] - 1) + iy*xlen + iz*xlen*ylen];
        val += nabt3*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        __syncthreads();
        block_sum = BlockReduce(temp_storage).Sum(val);
        if(threadId == 0) {
             atomicAdd(&d_res[1], block_sum);
        }
        
        tmp = psi_data[ix+(idy[iy + 1] - 1)*xlen + iz*xlen*ylen];
        val = nabt4*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        tmp = psi_data[ix + (idy[iy + 2] - 1)*xlen + iz*xlen*ylen];
        val += nabt5*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        tmp = psi_data[ix + (idy[iy + 3] - 1)*xlen + iz*xlen*ylen];
        val += nabt6*(cpsi.y*tmp.x+cpsi.x*tmp.y);
        tmp = psi_data[ix + (idy[iy + 4] - 1)*xlen + iz*xlen*ylen];
        val += nabt7*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        __syncthreads();
        block_sum = BlockReduce(temp_storage).Sum(val);
        if(threadId == 0) {
             atomicAdd(&d_res[2], block_sum);
        }
        
        tmp = psi_data[ix + iy*xlen + (idz[iz + 1] - 1)*xlen*ylen];
        val = nabt8*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        tmp = psi_data[ix + iy*xlen + (idz[iz + 2] - 1)*xlen*ylen];
        val += nabt9*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        tmp = psi_data[ix + iy*xlen + (idz[iz + 3] - 1)*xlen*ylen];
        val += nabt10*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        tmp = psi_data[ix + iy*xlen + (idz[iz + 4] - 1)*xlen*ylen];
        val += nabt11*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        __syncthreads();
        block_sum = BlockReduce(temp_storage).Sum(val);
        if(threadId == 0) {
             atomicAdd(&d_res[3], block_sum);
        }
    }

    void stencil_current_core_gpu(const int ik_s, const int ik_e, const int io_s, const int io_e, const double* const vec_k, const double* const vec_Ac,
                                  const int* const is_array, const int* const ie_array, const int* const is, const int* const ie, const int* const idx, const int* const idy,
                                  const int* const idz, const double* const nabt, const int ispin, const int im, const int spin_len, const hipDoubleComplex* const psi, 
                                  const double* const BT, const double* const rocc, const double* const wtk, double* const jx, double* const  jy, double* const jz) {
        const int xlen = ie_array[0] - is_array[0] + 1;
        const int ylen = ie_array[1] - is_array[1] + 1;
        const int zlen = ie_array[2] - is_array[2] + 1;
        const int xsize = ie[0] - is[0] + 1;
        const int ysize = ie[1] - is[1] + 1;
        const int zsize = ie[2] - is[2] + 1;
        const int maxlen = xsize*ysize*zsize;
        int psi_index = xlen*ylen*zlen*(ispin - 1) + xlen*ylen*zlen*spin_len*(ik_e - ik_s + 1)*(io_e - io_s + 1)*(im - 1);
        double res[4];
        double *d_nabt, *d_res;
        int *d_idx, *d_idy, *d_idz;

        hipMalloc(reinterpret_cast<void**>(&d_nabt), sizeof(double)*12);
        hipMalloc(reinterpret_cast<void**>(&d_idx), sizeof(int)*(ie[0] - is[0] + 1 + Nd));
        hipMalloc(reinterpret_cast<void**>(&d_idy), sizeof(int)*(ie[1] - is[1] + 1 + Nd));
        hipMalloc(reinterpret_cast<void**>(&d_idz), sizeof(int)*(ie[2] - is[2] + 1 + Nd));
        hipMalloc(reinterpret_cast<void**>(&d_res), sizeof(double)*4);

        hipMemcpy(d_nabt, nabt, sizeof(double)*12, hipMemcpyHostToDevice);
        hipMemcpy(d_idx, idx, sizeof(int)*(ie[0] - is[0] + 1 + Nd), hipMemcpyHostToDevice);
        hipMemcpy(d_idy, idy, sizeof(int)*(ie[1] - is[1] + 1 + Nd), hipMemcpyHostToDevice);
        hipMemcpy(d_idz, idz, sizeof(int)*(ie[2] - is[2] + 1 + Nd), hipMemcpyHostToDevice);
        for(int ik = ik_s - 1; ik < ik_e; ik++) {
            double kAc[3];
            for(int i = 0; i < 3; i++) {
                kAc[i] = vec_k[i + 3*ik] + vec_Ac[i];
            }
            for(int io = io_s - 1; io < io_e; io++) {
                const int grid_size = ((maxlen+block_size - 1)/block_size);
                // gpu kernel
                hipMemset(reinterpret_cast<void*>(d_res), static_cast<double>(0), sizeof(double)*4);
                stencil_current_kernel<<<grid_size, block_size, sizeof(double)*block_size>>>(
                        d_res, &psi[psi_index], xlen, ylen, xsize, ysize, zsize, maxlen, d_nabt, d_idx, d_idy, d_idz,
                        nabt[0], nabt[1], nabt[2], nabt[3], nabt[4], nabt[5], nabt[6], nabt[7], nabt[8], nabt[9], nabt[10], nabt[11]);
                hipMemcpy(res, d_res, sizeof(double)*4, hipMemcpyDeviceToHost);
                psi_index += xlen*ylen*zlen*spin_len;
                double wrk1[3], wrk2[3], wrk3[3], wrk4[3];
                for(int i = 0; i < 3; i++) {
                    wrk1[i] = kAc[i]*res[0];
                }
                wrk2[0] = res[1]*static_cast<double>(2);
                wrk2[1] = res[2]*static_cast<double>(2);
                wrk2[2] = res[3]*static_cast<double>(2);
                for(int i = 0; i < 3; i++) {
                    wrk3[i] = BT[i]*wrk2[0] + BT[i + 3]*wrk2[1] + BT[i + 6]*wrk2[2];
                }
                for(int i = 0; i < 3; i++) {
                    wrk4[i] = (wrk1[i] + wrk3[i])*rocc[io + ik*(io_e - io_s + 1)]*wtk[ik];
                }
                *jx += wrk4[0];
                *jy += wrk4[1];
                *jz += wrk4[2];
            }
        }
    }
}
