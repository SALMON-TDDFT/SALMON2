#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <complex>
#include <hip/hip_complex.h>
#include "array_index.h"
#include <thrust/device_vector.h>

extern "C" {
    static constexpr unsigned block_size = 128;
    static constexpr unsigned Nd = 4;
    static_assert(block_size != 0 && (block_size & (block_size-1)) == 0, "block_size must be 2^n");

    __device__ void thread_reduction(const int threadId, double* const d_res, double* const smem, const int blockDim) {
        for(int i = 1; i < blockDim; i *= 2) { // blockDim must be 2^n
            if ((threadId % (2*i)) == 0) {
                smem[threadId] += smem[threadId + i];
            }
            __syncthreads();
        }
        if(threadId == 0) {
            atomicAdd(d_res, smem[0]);
        }
    }

    __global__ void stencil_current_kernel(double* const d_res, const hipDoubleComplex* const psi_data, const int xlen,
                                           const int ylen, const int xoffset, const int yoffset, const int zoffset, const int xsize, const int ysize, const int zsize, const int maxlen,
                                           const double* const nabt, const int* const idx, const int* const idy, const int* const idz, const double nabt0, const double nabt1,
                                           const double nabt2, const double nabt3, const double nabt4, const double nabt5, const double nabt6, const double nabt7, const double nabt8,
                                           const double nabt9, const double nabt10, const double nabt11) {
        const int tid = blockIdx.x * blockDim.x + threadIdx.x;
        const int threadId = threadIdx.x;
        if (tid >= maxlen) {
            return;
        }
        extern __shared__ double smem[];
        const int ix = (tid % xsize) + xoffset;
        const int iy = ((tid/xsize) % ysize) + yoffset;
        const int iz = ((tid/(xsize*ysize)) % zsize) + zoffset;
        hipDoubleComplex tmp;
        const hipDoubleComplex cpsi = hipConj(psi_data[ARRAY_INDEX_3D(ix, iy, iz, 1, xlen, 1, ylen, 1)]);
        tmp = psi_data[ARRAY_INDEX_3D(ix, iy, iz, 1, xlen, 1, ylen, 1)]; // psi_data[ix][iy][iz]
        const double psi_abs = hipCabs(tmp);
        smem[threadId] = psi_abs*psi_abs;
        __syncthreads();
        thread_reduction(threadId, &d_res[0], smem, blockDim.x);

        tmp = psi_data[ARRAY_INDEX_3D(idx[ARRAY_INDEX_1D(ix+1, 1)], iy, iz, 1, xlen, 1, ylen, 1)];
        smem[threadId] = nabt0*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        tmp = psi_data[ARRAY_INDEX_3D(idx[ARRAY_INDEX_1D(ix+2, 1)], iy, iz, 1, xlen, 1, ylen, 1)];
        smem[threadId] += nabt1*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        tmp = psi_data[ARRAY_INDEX_3D(idx[ARRAY_INDEX_1D(ix+3, 1)], iy, iz, 1, xlen, 1, ylen, 1)];
        smem[threadId] += nabt2*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        tmp = psi_data[ARRAY_INDEX_3D(idx[ARRAY_INDEX_1D(ix+4, 1)], iy, iz, 1, xlen, 1, ylen, 1)];
        smem[threadId] += nabt3*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        __syncthreads();
        thread_reduction(threadId, &d_res[1], smem, blockDim.x);

        tmp = psi_data[ARRAY_INDEX_3D(ix, idy[ARRAY_INDEX_1D(iy+1, 1)], iz, 1, xlen, 1, ylen, 1)];
        smem[threadId] = nabt4*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        tmp = psi_data[ARRAY_INDEX_3D(ix, idy[ARRAY_INDEX_1D(iy+2, 1)], iz, 1, xlen, 1, ylen, 1)];
        smem[threadId] += nabt5*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        tmp = psi_data[ARRAY_INDEX_3D(ix, idy[ARRAY_INDEX_1D(iy+3, 1)], iz, 1, xlen, 1, ylen, 1)];
        smem[threadId] += nabt6*(cpsi.y*tmp.x+cpsi.x*tmp.y);
        tmp = psi_data[ARRAY_INDEX_3D(ix, idy[ARRAY_INDEX_1D(iy+4, 1)], iz, 1, xlen, 1, ylen, 1)];
        smem[threadId] += nabt7*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        __syncthreads();
        thread_reduction(threadId, &d_res[2], smem, blockDim.x);

        tmp = psi_data[ARRAY_INDEX_3D(ix, iy, idz[ARRAY_INDEX_1D(iz+1, 1)], 1, xlen, 1, ylen, 1)];
        smem[threadId] = nabt8*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        tmp = psi_data[ARRAY_INDEX_3D(ix, iy, idz[ARRAY_INDEX_1D(iz+2, 1)], 1, xlen, 1, ylen, 1)];
        smem[threadId] += nabt9*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        tmp = psi_data[ARRAY_INDEX_3D(ix, iy, idz[ARRAY_INDEX_1D(iz+3, 1)], 1, xlen, 1, ylen, 1)];
        smem[threadId] += nabt10*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        tmp = psi_data[ARRAY_INDEX_3D(ix, iy, idz[ARRAY_INDEX_1D(iz+4, 1)], 1, xlen, 1, ylen, 1)];
        smem[threadId] += nabt11*(cpsi.y*tmp.x + cpsi.x*tmp.y);
        __syncthreads();
        thread_reduction(threadId, &d_res[3], smem, blockDim.x);
    }

    void stencil_current_core_gpu(const int ik_s, const int ik_e, const int io_s, const int io_e, const double* const vec_k, const double* const vec_Ac,
                                  const int* const is_array, const int* const ie_array, const int* const is, const int* const ie, const int* const idx, const int* const idy,
                                  const int* const idz, const double* const nabt, const int ispin, const int im, const int spin_len, const hipDoubleComplex* const psi, 
                                  const double* const BT, const double* const rocc, const double* const wtk, double* const jx, double* const  jy, double* const jz) {
        const int xlen = ie_array[0] - is_array[0] + 1;
        const int ylen = ie_array[1] - is_array[1] + 1;
        const int zlen = ie_array[2] - is_array[2] + 1;
        const int xsize = ie[0] - is[0] + 1;
        const int ysize = ie[1] - is[1] + 1;
        const int zsize = ie[2] - is[2] + 1;
        const int maxlen = xsize*ysize*zsize;
        int psi_index = xlen*ylen*zlen*(ispin - 1) + xlen*ylen*zlen*spin_len*(ik_e - ik_s + 1)*(io_e - io_s + 1)*(im - 1);
        double res[4];
        double *d_nabt, *d_res;
        int *d_idx, *d_idy, *d_idz;

        hipMalloc(reinterpret_cast<void**>(&d_nabt), sizeof(double)*12);
        hipMalloc(reinterpret_cast<void**>(&d_idx), sizeof(int)*(ie[0] - is[0] + 1 + Nd));
        hipMalloc(reinterpret_cast<void**>(&d_idy), sizeof(int)*(ie[1] - is[1] + 1 + Nd));
        hipMalloc(reinterpret_cast<void**>(&d_idz), sizeof(int)*(ie[2] - is[2] + 1 + Nd));
        hipMalloc(reinterpret_cast<void**>(&d_res), sizeof(double)*4);

        hipMemcpy(d_nabt, nabt, sizeof(double)*12, hipMemcpyHostToDevice);
        hipMemcpy(d_idx, idx, sizeof(int)*(ie[0] - is[0] + 1 + Nd), hipMemcpyHostToDevice);
        hipMemcpy(d_idy, idy, sizeof(int)*(ie[1] - is[1] + 1 + Nd), hipMemcpyHostToDevice);
        hipMemcpy(d_idz, idz, sizeof(int)*(ie[2] - is[2] + 1 + Nd), hipMemcpyHostToDevice);
        for(int ik = ik_s; ik <= ik_e; ik++) {
            double kAc[3];
            for(int i = 1; i <= 3; i++) {
                kAc[ARRAY_INDEX_1D(i, 1)] = vec_k[ARRAY_INDEX_2D(i, ik, 1, 3, 1)] + vec_Ac[ARRAY_INDEX_1D(i, 1)];
            }
            for(int io = io_s; io <= io_e; io++) {
                const int grid_size = ((maxlen+block_size - 1)/block_size);
                // gpu kernel
                hipMemset(reinterpret_cast<void*>(d_res), static_cast<double>(0), sizeof(double)*4);
                stencil_current_kernel<<<grid_size, block_size, sizeof(double)*block_size>>>(
                        d_res, &psi[psi_index], xlen, ylen, is[0], is[1], is[2], xsize, ysize, zsize, maxlen, d_nabt, d_idx, d_idy, d_idz,
                        nabt[0], nabt[1], nabt[2], nabt[3], nabt[4], nabt[5], nabt[6], nabt[7], nabt[8], nabt[9], nabt[10], nabt[11]);
                hipMemcpy(res, d_res, sizeof(double)*4, hipMemcpyDeviceToHost);
                psi_index += xlen*ylen*zlen*spin_len;
                double wrk1[3], wrk2[3], wrk3[3], wrk4[3];
                for(int i = 0; i < 3; i++) {
                    wrk1[i] = kAc[i]*res[0];
                }
                wrk2[0] = res[1]*static_cast<double>(2);
                wrk2[1] = res[2]*static_cast<double>(2);
                wrk2[2] = res[3]*static_cast<double>(2);
                for(int i = 1; i <= 3; i++) {
                    wrk3[ARRAY_INDEX_1D(i, 1)] =
						BT[ARRAY_INDEX_2D(i, 1, 1, 3, 1)]*wrk2[ARRAY_INDEX_1D(1, 1)] +
						BT[ARRAY_INDEX_2D(i, 2, 1, 3, 1)]*wrk2[ARRAY_INDEX_1D(2, 1)] +
						BT[ARRAY_INDEX_2D(i, 3, 1, 3, 1)]*wrk2[ARRAY_INDEX_1D(3, 1)];
                }
                for(int i = 0; i < 3; i++) {
                    wrk4[i] = (wrk1[i] + wrk3[i])*rocc[ARRAY_INDEX_2D(io, ik, io_s, io_e, 1)]*wtk[ARRAY_INDEX_1D(ik, 1)];
                }
                *jx += wrk4[ARRAY_INDEX_1D(1, 1)];
                *jy += wrk4[ARRAY_INDEX_1D(2, 1)];
                *jz += wrk4[ARRAY_INDEX_1D(3, 1)];
            }
        }
    }
}
