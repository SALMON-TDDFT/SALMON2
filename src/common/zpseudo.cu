
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#define ARRAY_INDEX_1D(C_i, Fort_i_start) ((C_i) - (Fort_i_start))
#define ARRAY_INDEX_3D(C_i, C_j, C_k, Fort_i_start, Fort_i_end, Fort_j_start, Fort_j_end, Fort_k_start, Fort_k_end) \
	(((C_i) - (Fort_i_start)) \
	 + ((C_j) - (Fort_j_start)) * ((Fort_i_end) - (Fort_i_start) + 1) \
	 + ((C_j) - (Fort_j_start)) * ((Fort_i_end) - (Fort_i_start) + 1) * ((Fort_j_end) - (Fort_j_start) + 1))
#define ARRAY_INDEX_7D(C0, C1, C2, C3, C4, C5, C6, F0s, F0e, F1s, F1e, F2s, F2e, F3s, F3e, F4s, F4e, F5s, F5e, F6s, F6e) \
	(((C0) - (F0s)) \
	+ ((C1) - (F1s)) * ((F0e) - (F0s) + 1) \
	+ ((C2) - (F2s)) * ((F0e) - (F0s) + 1) * ((F1e) - (F1s) + 1) \
	+ ((C3) - (F3s)) * ((F0e) - (F0s) + 1) * ((F1e) - (F1s) + 1)  * ((F2e) - (F2s) + 1) \
	+ ((C4) - (F4s)) * ((F0e) - (F0s) + 1) * ((F1e) - (F1s) + 1)  * ((F2e) - (F2s) + 1) * ((F3e) - (F3s) + 1) \
	+ ((C5) - (F5s)) * ((F0e) - (F0s) + 1) * ((F1e) - (F1s) + 1)  * ((F2e) - (F2s) + 1) * ((F3e) - (F3s) + 1) * ((F4e) - (F4s) + 1) \
	+ ((C6) - (F6s)) * ((F0e) - (F0s) + 1) * ((F1e) - (F1s) + 1)  * ((F2e) - (F2s) + 1) * ((F3e) - (F3s) + 1) * ((F4e) - (F4s) + 1) * ((F5e) - (F5s) + 1) \
	 )

__device__ hipDoubleComplex operator*(const hipDoubleComplex& a, const hipDoubleComplex& b) {
	return make_double2(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);
}

__device__ hipDoubleComplex operator*=(hipDoubleComplex& a, const double b) {
	return a = make_double2(a.x * b, a.y * b);
}

__device__ hipDoubleComplex operator+=(hipDoubleComplex& a, const hipDoubleComplex& b) {
	return a = make_double2(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);
}

// Kernel function for (src/common/nonlocal_potential.f90: l.271)
// Num threads = (im_e - im_s + 1) * (ik_e - ik_s + 1) * (io_e - io_s + 1) * Nlma
__global__ void zpseudo_kernel(
		// Output & Input
		// allocate(psi%zwf(mg%is_array(1):mg%ie_array(1),  &
        //           mg%is_array(2):mg%ie_array(2),  &
        //           mg%is_array(3):mg%ie_array(3),  &
        //           nspin,info%io_s:info%io_e,info%ik_s:info%ik_e,info%im_s:info%im_e))
		hipDoubleComplex* const htpsi_zwf,
		// Input
		const unsigned im_s,
		const unsigned im_e,
		const unsigned ik_s,
		const unsigned ik_e,
		const unsigned io_s,
		const unsigned io_e,
		const unsigned Nspin,
		const unsigned iz_min,
		const unsigned iz_max,
		const unsigned iy_min,
		const unsigned iy_max,
		const unsigned ix_min,
		const unsigned ix_max,
		const unsigned Nlma,
		const unsigned ppg_nps,
		const unsigned natom,
		const unsigned mg_is_array_1,
		const unsigned mg_ie_array_1,
		const unsigned mg_is_array_2,
		const unsigned mg_ie_array_2,
		const unsigned mg_is_array_3,
		const unsigned mg_ie_array_3,
		// allocate(ppg%ia_tbl(n*natom))
		const unsigned* const ppg_ia_tbl,
		// allocate(ppg%mps(natom))
		const unsigned* const ppg_mps,
		// allocate(ppg%jxyz(3,ppg%nps,natom))
		const unsigned* const ppg_jxyz,
		// allocate(ppg%zekr_uV(ppg%nps,ppg%nlma,ik_s:ik_e))
		const hipDoubleComplex* const ppg_zekr_uV,
		// allocate(ppg%rinv_uvu(n*natom))
		const double* const ppg_rinv_uvu,
		// Same as htpsi_zwf
		hipDoubleComplex* const tpsi_zwf
		) {
	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;


	const unsigned im_size = im_e - im_s + 1;
	const unsigned ik_size = ik_e - ik_s + 1;
	const unsigned io_size = io_e - io_s + 1;
	const unsigned array_length = im_size * ik_size * io_size * Nspin;
	if (tid >= array_length) {
		return;
	}

	const unsigned im = tid % im_size;
	const unsigned ik = (tid / im_size) % ik_size;
	const unsigned io = (tid / (im_size * ik_size)) % io_size;
	const unsigned ispin = (tid / (im_size * ik_size * io_size));

	for (unsigned ilma = 0; ilma < Nlma; ilma++) {
		const unsigned ia = ppg_ia_tbl[ilma];
		hipDoubleComplex uVpsi = make_double2(0., 0.);

		for (unsigned j = 0; j < ppg_mps[ARRAY_INDEX_1D(ia, 1)]; j++) {
			const unsigned ix = ppg_jxyz[ARRAY_INDEX_3D(1, j, ia, 1, 3, 1, ppg_nps, 1, natoms)];
			const unsigned iy = ppg_jxyz[ARRAY_INDEX_3D(2, j, ia, 1, 3, 1, ppg_nps, 1, natoms)];
			const unsigned iz = ppg_jxyz[ARRAY_INDEX_3D(3, j, ia, 1, 3, 1, ppg_nps, 1, natoms)];

			const hipDoubleComplex ppg_zekr_uV_v = ppg_zekr_uV[ARRAY_INDEX_3D(j, ilma, ik, 1, ppg_nps, 1, Nlma, ik_s, ik_e)];
			const hipDoubleComplex conjg_ppg_zekr_uV = make_double2(ppg_zekr_uV_v.x, -ppg_zekr_uV_v.y);
			uVpsi += conjg_ppg_zekr_uV * tpsi_zwf[ARRAY_INDEX_7D(
					ix, iy, iz, ispin, io, ik, im,
					mg_is_array_1, mg_ie_array_1,
					mg_is_array_2, mg_ie_array_2,
					mg_is_array_3, mg_ie_array_3,
					1, Nspin,
					io_s, io_e,
					ik_s, ik_e,
					im_s, im_e
					)];
		}

		uVpsi *= ppg_rinv_uvu[ARRAY_INDEX_1D(ia, 1)];

		for (unsigned j = 0; j < ppg_mps[ARRAY_INDEX_1D(ia, 1)]; j++) {
			const unsigned ix = ppg_jxyz[ARRAY_INDEX_3D(1, j, ia, 1, 3, 1, ppg_nps, 1, natoms)];
			const unsigned iy = ppg_jxyz[ARRAY_INDEX_3D(2, j, ia, 1, 3, 1, ppg_nps, 1, natoms)];
			const unsigned iz = ppg_jxyz[ARRAY_INDEX_3D(3, j, ia, 1, 3, 1, ppg_nps, 1, natoms)];

			const hipDoubleComplex wrk = uVpsi * ppg_zekr_uV[ARRAY_INDEX_3D(j, ilma, ik, 1, ppg_nps, 1, Nlma, ik_s, ik_e)];
			htpsi_zwf[ARRAY_INDEX_7D(
					ix, iy, iz, ispin, io, ik, im,
					mg_is_array_1, mg_ie_array_1,
					mg_is_array_2, mg_ie_array_2,
					mg_is_array_3, mg_ie_array_3,
					1, Nspin,
					io_s, io_e,
					ik_s, ik_e,
					im_s, im_e
					)] += wrk;
		}
	}
}


void zpseudo(
		// Output & Input
		hipDoubleComplex* const htpsi_zwf,
		// Input
		const unsigned im_s,
		const unsigned im_e,
		const unsigned ik_s,
		const unsigned ik_e,
		const unsigned io_s,
		const unsigned io_e,
		const unsigned Nspin,
		const unsigned iz_min,
		const unsigned iz_max,
		const unsigned iy_min,
		const unsigned iy_max,
		const unsigned ix_min,
		const unsigned ix_max,
		const unsigned Nlma,
		const unsigned ppg_nps,
		const unsigned natom,
		const unsigned mg_is_array_1,
		const unsigned mg_ie_array_1,
		const unsigned mg_is_array_2,
		const unsigned mg_ie_array_2,
		const unsigned mg_is_array_3,
		const unsigned mg_ie_array_3,
		const unsigned* const ppg_ia_tbl,
		const unsigned* const ppg_mps,
		const unsigned* const ppg_jxyz,
		const hipDoubleComplex* const ppg_zekr_uV,
		const double* const ppg_rinv_uvu,
		hipDoubleComplex* const tpsi_zwf
		) {
	const unsigned im_size = im_e - im_s + 1;
	const unsigned ik_size = ik_e - ik_s + 1;
	const unsigned io_size = io_e - io_s + 1;
	const unsigned num_threads = im_size * ik_size * io_size * Nspin;

	const unsigned block_size = 256;
	const unsigned grid_size = (num_threads + block_size - 1) / block_size;

	zpseudo_kernel<<<grid_size, block_size>>>(
		htpsi_zwf,
		// Input
		im_s,
		im_e,
		ik_s,
		ik_e,
		io_s,
		io_e,
		Nspin,
		iz_min,
		iz_max,
		iy_min,
		iy_max,
		ix_min,
		ix_max,
		Nlma,
		ppg_nps,
		natom,
		mg_is_array_1,
		mg_ie_array_1,
		mg_is_array_2,
		mg_ie_array_2,
		mg_is_array_3,
		mg_ie_array_3,
		ppg_ia_tbl,
		ppg_mps,
		ppg_jxyz,
		ppg_zekr_uV,
		ppg_rinv_uvu,
		tpsi_zwf
		);
	hipDeviceSynchronize();
}
