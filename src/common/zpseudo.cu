#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include "array_index.h"

extern "C" {
__host__ __device__ hipDoubleComplex operator*( const hipDoubleComplex& a
                                             , const hipDoubleComplex& b
)
{
  return make_double2(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);
}

__host__ __device__ hipDoubleComplex operator*=( hipDoubleComplex& a
                                              , const double b
)
{
  return a = make_double2(a.x * b, a.y * b);
}

__host__ __device__ hipDoubleComplex operator+=( hipDoubleComplex& a
                                              , const hipDoubleComplex& b
)
{
  return a = make_double2(a.x + b.x, a.y + b.y);
}

// Kernel function for (zpseudo in src/common/nonlocal_potential.f90)
// Num threads = (im_e - im_s + 1) * (ik_e - ik_s + 1) * (io_e - io_s + 1) * Nlma.
__global__ void zpseudo_kernel( hipDoubleComplex* const htpsi_zwf
                              // Shape :  (psi%zwf(mg%is_array(1):mg%ie_array(1),  &
                              //           mg%is_array(2):mg%ie_array(2),  &
                              //           mg%is_array(3):mg%ie_array(3),  &
                              //           nspin,info%io_s:info%io_e,info%ik_s:info%ik_e,info%im_s:info%im_e))
                              //
                              // Input
                              , const int im_s
                              , const int im_e
                              , const int ik_s
                              , const int ik_e
                              , const int io_s
                              , const int io_e
                              , const int Nspin
                              , const int Nlma
                              , const int ppg_nps
                              , const int natom
                              , const int mg_is_array_1
                              , const int mg_ie_array_1
                              , const int mg_is_array_2
                              , const int mg_ie_array_2
                              , const int mg_is_array_3
                              , const int mg_ie_array_3
                              , const int* const ppg_ia_tbl
                              // Shape :  (ppg%ia_tbl(n*natom))
                              , const int* const ppg_mps
                              // Shape :  (ppg%mps(natom))
                              , const int* const ppg_jxyz
                              // Shape :  (ppg%jxyz(3,ppg%nps,natom))
                              , const hipDoubleComplex* const ppg_zekr_uV
                              // Shape :  (ppg%zekr_uV(ppg%nps,ppg%nlma,ik_s:ik_e))
                              , const double* const ppg_rinv_uvu
                              // Shape :  (ppg%rinv_uvu(n*natom))
                              , const hipDoubleComplex* const tpsi_zwf
                              // Shape :  The same with htpsi_zwf
)
{
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;


  const unsigned im_size = im_e - im_s + 1;
  const unsigned ik_size = ik_e - ik_s + 1;
  const unsigned io_size = io_e - io_s + 1;
  const unsigned array_length = im_size * ik_size * io_size * Nspin * Nlma;
  if (tid >= array_length)
  {
    return;
  }

  const unsigned im = tid % im_size + im_s;
  const unsigned ik = (tid / im_size) % ik_size + ik_s;
  const unsigned io = (tid / (im_size * ik_size)) % io_size + io_s;
  const unsigned ispin = (tid / (im_size * ik_size * io_size)) % Nspin+ 1;
  const unsigned ilma = (tid / (im_size * ik_size * io_size * Nspin)) + 1;

  {
    const unsigned ia = ppg_ia_tbl[ARRAY_INDEX_1D(ilma, 1)];
    hipDoubleComplex uVpsi = make_double2(0., 0.);

    for (unsigned j = 1; j <= ppg_mps[ARRAY_INDEX_1D(ia, 1)]; j++)
    {
      const hipDoubleComplex ppg_zekr_uV_v = ppg_zekr_uV[ARRAY_INDEX_3D(j, ilma, ik, 1, ppg_nps, 1, Nlma, ik_s)];
      // calculate conj
      const hipDoubleComplex conjg_ppg_zekr_uV = make_double2(ppg_zekr_uV_v.x, -ppg_zekr_uV_v.y);

      const unsigned ix = ppg_jxyz[ARRAY_INDEX_3D(1, j, ia, 1, 3, 1, ppg_nps, 1)];
      const unsigned iy = ppg_jxyz[ARRAY_INDEX_3D(2, j, ia, 1, 3, 1, ppg_nps, 1)];
      const unsigned iz = ppg_jxyz[ARRAY_INDEX_3D(3, j, ia, 1, 3, 1, ppg_nps, 1)];
      uVpsi += conjg_ppg_zekr_uV * tpsi_zwf[ARRAY_INDEX_7D( ix, iy, iz, ispin, io, ik, im
                                                          , mg_is_array_1, mg_ie_array_1
                                                          , mg_is_array_2, mg_ie_array_2
                                                          , mg_is_array_3, mg_ie_array_3
                                                          , 1, Nspin
                                                          , io_s, io_e
                                                          , ik_s, ik_e
                                                          , im_s
                                                          )];
    }

    uVpsi *= ppg_rinv_uvu[ARRAY_INDEX_1D(ilma, 1)];

    for (unsigned j = 1; j <= ppg_mps[ARRAY_INDEX_1D(ia, 1)]; j++)
    {
      const hipDoubleComplex wrk = uVpsi * ppg_zekr_uV[ARRAY_INDEX_3D(j, ilma, ik, 1, ppg_nps, 1, Nlma, ik_s)];

      const unsigned ix = ppg_jxyz[ARRAY_INDEX_3D(1, j, ia, 1, 3, 1, ppg_nps, 1)];
      const unsigned iy = ppg_jxyz[ARRAY_INDEX_3D(2, j, ia, 1, 3, 1, ppg_nps, 1)];
      const unsigned iz = ppg_jxyz[ARRAY_INDEX_3D(3, j, ia, 1, 3, 1, ppg_nps, 1)];

      const unsigned mem_offset = ARRAY_INDEX_7D( ix, iy, iz, ispin, io, ik, im
                                                , mg_is_array_1, mg_ie_array_1
                                                , mg_is_array_2, mg_ie_array_2
                                                , mg_is_array_3, mg_ie_array_3
                                                , 1, Nspin
                                                , io_s, io_e
                                                , ik_s, ik_e
                                                , im_s
                                                );
      atomicAdd(&(htpsi_zwf[mem_offset].x), wrk.x);
      atomicAdd(&(htpsi_zwf[mem_offset].y), wrk.y);
    }
  }
}

void zpseudo_cuda( hipDoubleComplex* const htpsi_zwf
                 , const int n
                 , const int im_s
                 , const int im_e
                 , const int ik_s
                 , const int ik_e
                 , const int io_s
                 , const int io_e
                 , const int Nspin
                 , const int Nlma
                 , const int ppg_nps
                 , const int natom
                 , const int mg_is_array_1
                 , const int mg_ie_array_1
                 , const int mg_is_array_2
                 , const int mg_ie_array_2
                 , const int mg_is_array_3
                 , const int mg_ie_array_3
                 , const int* const ppg_ia_tbl
                 , const int* const ppg_mps
                 , const int* const ppg_jxyz
                 , const hipDoubleComplex* const ppg_zekr_uV
                 , const double* const ppg_rinv_uvu
                 , hipDoubleComplex* const tpsi_zwf
)
{
  const unsigned im_size = im_e - im_s + 1;
  const unsigned ik_size = ik_e - ik_s + 1;
  const unsigned io_size = io_e - io_s + 1;
  const unsigned num_threads = im_size * ik_size * io_size * Nspin * Nlma;

  constexpr unsigned block_size = 256;
  const unsigned grid_size = (num_threads + block_size - 1) / block_size;

  zpseudo_kernel<<<grid_size, block_size>>>( htpsi_zwf
                                           , im_s
                                           , im_e
                                           , ik_s
                                           , ik_e
                                           , io_s
                                           , io_e
                                           , Nspin
                                           , Nlma
                                           , ppg_nps
                                           , natom
                                           , mg_is_array_1
                                           , mg_ie_array_1
                                           , mg_is_array_2
                                           , mg_ie_array_2
                                           , mg_is_array_3
                                           , mg_ie_array_3
                                           , ppg_ia_tbl
                                           , ppg_mps
                                           , ppg_jxyz
                                           , ppg_zekr_uV
                                           , ppg_rinv_uvu
                                           , tpsi_zwf
                                           );
  hipDeviceSynchronize();
}
} // extern "C"
